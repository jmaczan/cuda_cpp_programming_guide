
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int N = 1000000;
    size_t size = N * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i;
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("C[1] = %f\n", h_C[1]);

    free(h_A);
    free(h_B);
    free(h_C);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}